#include "hip/hip_runtime.h"
#include <vector>
#include "gtest/gtest.h"
#include "rxmesh/util/cuda_query.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/report.h"

struct RXMeshTestArg
{
    uint32_t    num_run       = 1;
    uint32_t    device_id     = 0;
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    int         argc          = argc;
    char**      argv          = argv;
} rxmesh_args;

// clang-format off
#include "test_queries.h"
#include "test_patch_scheduler.cuh"
#include "test_patch_lock.cuh"
#include "test_wasted_work.cuh"
#include "test_grad.h"
// clang-format on

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    rxmesh_args.argc = argc;
    rxmesh_args.argv = argv;
    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: RXMesh_test.exe < -option X>\n"
                        " -h:          Display this massage and exit\n"
                        " -input:      Input OBJ mesh file. Default is {} \n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -num_run:    Number of iterations for performance testing. Default is {} \n"
                        " -device_id:  GPU device ID. Default is {}",
            rxmesh_args.obj_file_name, rxmesh_args.output_folder ,rxmesh_args.num_run,rxmesh_args.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }


        if (cmd_option_exists(argv, argc + argv, "-num_run")) {
            rxmesh_args.num_run =
                atoi(get_cmd_option(argv, argv + argc, "-num_run"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            rxmesh_args.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            rxmesh_args.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            rxmesh_args.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }


    RXMESH_INFO("input= {}", rxmesh_args.obj_file_name);
    RXMESH_INFO("output_folder= {}", rxmesh_args.output_folder);
    RXMESH_INFO("num_run= {}", rxmesh_args.num_run);
    RXMESH_INFO("device_id= {}", rxmesh_args.device_id);

    cuda_query(rxmesh_args.device_id);

    return RUN_ALL_TESTS();
}
