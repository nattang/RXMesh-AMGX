#include "hip/hip_runtime.h"
// Parallel version of
// Fleishman, Shachar, Iddo Drori, and Daniel Cohen-Or.
//"Bilateral mesh denoising." ACM SIGGRAPH 2003 Papers.2003. 950-953.

#include <omp.h>

#include "../common/openmesh_trimesh.h"
#include "gtest/gtest.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/export_tools.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/log.h"

struct arg
{
    std::string obj_file_name   = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder   = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id       = 0;
    uint32_t    num_filter_iter = 5;
    char**      argv;
    int         argc;
} Arg;

#include "filtering_openmesh.h"
#include "filtering_rxmesh.cuh"

TEST(App, Filtering)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);


    TriMesh input_mesh;
    ASSERT_TRUE(OpenMesh::IO::read_mesh(input_mesh, Arg.obj_file_name));

    // OpenMesh Impl
    std::vector<std::vector<dataT>> ground_truth(input_mesh.n_vertices());
    for (auto& g : ground_truth) {
        g.resize(3);
    }
    size_t                          max_neighbour_size = 0;
    filtering_openmesh<dataT>(
        omp_get_max_threads(), input_mesh, ground_truth, max_neighbour_size);


    // RXMesh Impl
    filtering_rxmesh<dataT>(
        Arg.obj_file_name, ground_truth, max_neighbour_size);
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;

    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: Filtering.exe < -option X>\n"
                        " -h:                Display this massage and exit\n"
                        " -input:      Input OBJ mesh file. Default is {} \n"
                        " -o:                JSON file output folder. Default is {} \n"
                        " -num_filter_iter:  Iteration count. Default is {} \n"
                        " -device_id:        GPU device ID. Default is {}",
             Arg.obj_file_name, Arg.output_folder ,Arg.num_filter_iter ,Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-num_filter_iter")) {
            Arg.num_filter_iter =
                atoi(get_cmd_option(argv, argv + argc, "-num_filter_iter"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("num_filter_iter= {}", Arg.num_filter_iter);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}